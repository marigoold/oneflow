#include "hip/hip_runtime.h"
/*
Copyright 2020 The OneFlow Authors. All rights reserved.

Licensed under the Apache License, Version 2.0 (the "License");
you may not use this file except in compliance with the License.
You may obtain a copy of the License at

    http://www.apache.org/licenses/LICENSE-2.0

Unless required by applicable law or agreed to in writing, software
distributed under the License is distributed on an "AS IS" BASIS,
WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
See the License for the specific language governing permissions and
limitations under the License.
*/
#include "oneflow/user/kernels/embedding_kernel_util.h"
#include <cstdint>

namespace oneflow {

namespace {

template<typename T, typename index_T>
__global__ void embedding_kernel(const T* weight_buf, const index_T* indices_buf, T* out_buf, 
                                 const int32_t num_indices,const int32_t emb_dim) {
  CUDA_1D_KERNEL_LOOP(i, num_indices * emb_dim){
     int32_t indices_index = i / emb_dim;
     int32_t dim_index = i - i * indices_index;
     int32_t from_index = indices_buf[indices_index] + dim_index;
     out_buf[i] = weight_buf[from_index];
  }
}

template<typename T, typename index_T>
__global__ void embedding_grad_kernel(const T* dy_buf, const index_T* indices_buf, T* dx_buf, const int32_t padding_idx,
                                      const int32_t num_indices, const int32_t emb_dim) {
       CUDA_1D_KERNEL_LOOP(i, num_indices * emb_dim){
           int32_t indice = indices_buf[i];
           int32_t indices_index = i / emb_dim;
           int32_t dim_index = i - i * indices_index;
           int32_t from_index = indices_buf[indices_index] + dim_index;
           if(indice != padding_idx){
               dx_buf[from_index] += dy_buf[i];
           }
       }
  }
}

template<typename index_T>
__global__ void indices_freq(const index_T* indices_buf, const int32_t num_indices, index_T * tmp_buf){
        CUDA_1D_KERNEL_LOOP(i, num_indices){
           tmp_buf[indices_buf[i]]++;
        }
}

template<typename T, typename index_T>
__global__ void embedding_scale(T* dx_buf, const int32_t emb_size, const int32_t emb_dim, index_T * tmp_buf){
        CUDA_1D_KERNEL_LOOP(i, emb_size*emb_dim){
            int32_t emb_size_index = i/emb_dim;
            if(tmp_buf[i]>1){
                dx_buf[i]/=tmp_buf[emb_size_index];
            }
        }
}

}



template<typename T, typename index_T>
struct EmbeddingFunctor<DeviceType::kCUDA, T, index_T> final{
    void operator()(ep::Stream* stream, const T* weight_buf, const index_T* indices_buf, T* out_buf,
                    const int32_t padding_idx, const bool scale_grad_by_freq,  
                    const int32_t num_indices, const int32_t emb_dim, const int32_t emb_size){
        embedding_kernel<T, index_T>
          <<<BlocksNum4ThreadsNum(num_indices * emb_dim), kCudaThreadsNumPerBlock, 0,
             stream->As<ep::CudaStream>()->hip_stream()>>>(weight_buf, indices_buf, out_buf, num_indices, emb_dim);
    }
};


template<typename T, typename index_T>
struct EmbeddingGradFunctor<DeviceType::kCUDA, T, index_T> final{
    void operator()(ep::Stream* stream, const T* dy_buf, const index_T* indices_buf, T* dx_buf,
                    const int32_t padding_idx, const bool scale_grad_by_freq,  const int32_t num_indices, const int32_t emb_dim, const int32_t emb_size
                    index_T * tmp_buf){

        embedding_grad_kernel<T, index_T>
          <<<BlocksNum4ThreadsNum(num_indices*emb_dim), kCudaThreadsNumPerBlock, 0, 
          stream->As<ep::CudaStream>()->hip_stream()>>>(dy_buf, indices_buf, dx_buf, padding_idx, num_indices, emb_dim);
        
        if(scale_grad_by_freq){
           indices_freq<index_T><<<BlocksNum4ThreadsNum(num_indices), kCudaThreadsNumPerBlock, 0,
           stream->As<ep::CudaStream>()->hip_stream()>>>(indices_buf, num_indices, tmp_buf);

           embedding_scale<T, index_T><<<BlocksNum4ThreadsNum(emb_size * emb_dim), kCudaThreadsNumPerBlock, 0,
           stream->As<ep::CudaStream>()->hip_stream()>>>(dx_buf, emb_size, emb_dim, tmp_buf);
        }

        ////
        for(int i=0;i<num_indices*emb_dim;i++){
           int32_t indice = indices_buf[i];
           int32_t indices_index = i / emb_dim;
           int32_t dim_index = i - i * indices_index;
           int32_t from_index = indices_buf[indices_index] + dim_index;
           if(indice != padding_idx){
               dx[from_index] += dy_buf[i]
           }
        }
        
        for(int32_t i = 0; i < num_indices; i++){
            tmp_buf[indices_buf[i]]++;
        }

        for(int i=0;i<emb_size*emb_dim;i++){
            int32_t emb_size_index = i/emb_dim;
            if(tmp_buf[i]>1){
                dx_buf[i]/=tmp_buf[emb_size_index];
            }
        }
        ////
        
        for(int32_t i = 0;i < num_indices; i++){
            int32_t indice = indices_buf[i];
            if(indice != padding_idx){
                 const T* from = dy_buf + i * emb_dim;
                 T* to = dx_buf + indice * emb_dim;
                 std::transform(from, from + emb_dim, to, to, std::plus<T>());
            }
        }

        if(scale_grad_by_freq){
            std::vector<index_T> indice_freq(emb_size, 0);
            for(int32_t i = 0; i < num_indices; i++){
                indice_freq[indices_buf[i]]++;
            }

            for(int32_t i = 0; i< emb_size;i++){
                if(indice_freq[i]>1){
                    T* from = dx_buf + i * emb_dim;
                    for(int32_t j=0; j<emb_dim; j++){
                        from[j]/=indice_freq[i];
                    }
                }
            }
        }
        
    }
};

#define INITIATE_EMBEDDING_KERNEL_UTIL_CPU_IMPL(in_type_pair, index_type_pair)                   \
      template struct EmbeddingRenormFunctor<DeviceType::kCPU, OF_PP_PAIR_FIRST(in_type_pair),   \
                                       OF_PP_PAIR_FIRST(index_type_pair)>;                       \
      template struct EmbeddingFunctor<DeviceType::kCPU, OF_PP_PAIR_FIRST(in_type_pair),         \
                                       OF_PP_PAIR_FIRST(index_type_pair)>;                       \
      template struct EmbeddingGradFunctor<DeviceType::kCPU, OF_PP_PAIR_FIRST(in_type_pair),     \
                                       OF_PP_PAIR_FIRST(index_type_pair)>;
OF_PP_SEQ_PRODUCT_FOR_EACH_TUPLE(INITIATE_EMBEDDING_KERNEL_UTIL_CPU_IMPL, EMBEDDING_DATA_TYPE_SEQ,
                                 INDEX_DATA_TYPE_SEQ);
#undef INITIATE_EMBEDDING_KERNEL_UTIL_CPU_IMPL

}  // namespace oneflow